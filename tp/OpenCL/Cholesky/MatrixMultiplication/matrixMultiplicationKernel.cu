#ifndef MATRIXMULTIPLICATIONKERNEL_CU
#define MATRIXMULTIPLICATIONKERNEL_CU

#include <hiprand/hiprand.h>


__global__ void matrixMul(float * g_C, float * g_A, float *g_B,int wa, int wb){


  int x = blockIdx.x * blockDim.y + threadIdx.x;
  int y = blockIdx.y*blockDim.x + threadIdx.y;
  
  float result = 0;

  int i = 0;
  for(i = 0; i < wa; ++i){
    float tempA = g_A[y*wa+i];
    float tempB = g_B[i*wb+x];
    result += tempA*tempB;
  }

  g_C[y*wa+x] = result;

}


__global__ void matrixMul2(float * g_C, float * g_A, float *g_B,int wa, int wb){
  int TILE_SIZE = 16;

  int x = blockIdx.x*TILE_SIZE + threadIdx.x;
  int y = blockIdx.y*TILE_SIZE + threadIdx.y;
  
  float result = 0;

  int i = 0;
  for(i = 0; i < wa; ++i){
    float tempA = g_A[y*wa+i];
    float tempB = g_B[i*wb+x];
    result += tempA*tempB;
  }

  g_C[y*wa+x] = result;

}



__global__ void matrixMul3(float * g_C, float * g_A, float *g_B,int wa, int wb){
  const int TILE_WIDTH = 16;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int bx = blockIdx.x;
  int by = blockIdx.y;

  __shared__ float s_a[TILE_WIDTH][TILE_WIDTH];
  __shared__ float s_b[TILE_WIDTH][TILE_WIDTH];

  int row = bx*blockDim.y + tx;
  int col = by*blockDim.x + ty;



  float result = 0;

  int i = 0;
  for(i = 0; i < wa/TILE_WIDTH; ++i){
    s_a[tx][ty] = g_A[i*TILE_WIDTH + row*wa +ty];
    s_b[tx][ty] = g_B[(i*TILE_WIDTH*wa)+tx*wa+ col];
    __syncthreads();

    int k =0;
    for(k=0;k<TILE_WIDTH;++k){
      result += s_a[tx][k] * s_b[k][ty];
    }
    __syncthreads();
  }

  g_C[row*wa+col] = result;

}



#endif
