#include "matrixMultiplication.h"

#include "matrixMultiplicationKernel.cu"



#define WA 2048
#define HA WA
#define WB WA
#define HB WA
#define WC WA
#define HC HB

#define BLOCK_SIZE 16



int main(int argc, char ** argv){
  double performance;
  perf_t start,stop;

  long flop = (long)2*WA*WA*WA;


  // C = A * B

  srand(2012);

  // Print informaitons about the system
  print_information();

  // Allocation Memmory for matrix
  float * A = (float *) malloc(sizeof(float)*WA*HA);
  float * B = (float *) malloc(sizeof(float)*WB*HB);
  float * C = (float *) malloc(sizeof(float)*WC*HC);
  float * D = (float *) malloc(sizeof(float)*WC*HC);
  memset(C,0,WC*HC*sizeof(float));
  memset(D,0,WC*HC*sizeof(float));

  // Init matrix
  init_matrix(A,WA,HA);
  init_matrix(B,WB,HB);

  struct timeval tv;
  double time_start;
  double time_end;
  
  double time_gpu;
  double time_cpu;

  // Start Multiplication CPU

  printf("\n\nMultiplication with CPU using openMP\n");
  printf("-----------------------------------\n");

  printf("\nSize Matrix %dx%d\n",WA,WB);



  gettimeofday(&tv,NULL);
  time_start = tv.tv_sec+(tv.tv_usec/1000000.0);
  printf("Starting multiplication CPU ...");
  perf(&start);
  matrixMultiplicationOMP(A,WA,HA,B,WB,HB,D,WC,HC);
  perf(&stop);
  gettimeofday(&tv,NULL);
  time_end = tv.tv_sec+(tv.tv_usec/1000000.0);
  printf("[OK]\n");
  printf("Time : %f s\n",(float)(time_end-time_start));
  perf_diff(&start, &stop);
  performance = perf_mflops(&stop, flop); 
  printf("Mflop/s : %lf \n", performance);
  time_cpu = time_end-time_start;
 

  printf("\n\nMultiplication without optimisation\n");
  printf("-----------------------------------\n");

  printf("\nSize Matrix %dx%d\n",WA,WB);

  

  // Start Multiplication GPU

  gettimeofday(&tv,NULL);
  time_start = tv.tv_sec+(tv.tv_usec/1000000.0);
  printf("\nStarting multiplication GPU ...");
  perf(&start);
  matrixMultiplication(A,WA,HA,B,WB,HB,C,WC,HC);
  perf(&stop);
  gettimeofday(&tv,NULL);
  time_end = tv.tv_sec+(tv.tv_usec/1000000.0);
  printf("[OK]\n");
  printf("Time : %f s\n",(float)(time_end-time_start));

  perf_diff(&start, &stop);
  performance = perf_mflops(&stop, flop); 
  printf("Mflop/s : %lf \n", performance);


  time_gpu = time_end-time_start;
  

  if(matrix_compare(C,D,WC*HC)){
    printf("Multiplication correct\n");
  }
  else 
    printf("Multiplication incorrect\n");

  int speed_up = (int)time_cpu/time_gpu;
  printf("Speed UP x%d\n",speed_up);

  memset(C,0,WC*HC*sizeof(float));

  printf("\n\nMultiplication with first optimisation\n");
  printf("-----------------------------------------\n");

  printf("\nSize Matrix %dx%d\n",WA,WB);

  // Start Multiplication GPU

  gettimeofday(&tv,NULL);
  time_start = tv.tv_sec+(tv.tv_usec/1000000.0);
  printf("Starting multiplication GPU ...");
  perf(&start);
  matrixMultiplication2(A,WA,HA,B,WB,HB,C,WC,HC);
  perf(&stop);
  gettimeofday(&tv,NULL);
  time_end = tv.tv_sec+(tv.tv_usec/1000000.0);
  printf("[OK]\n");
  printf("Time : %f s\n",(float)(time_end-time_start));

  perf_diff(&start, &stop);
  performance = perf_mflops(&stop, flop); 
  printf("Mflop/s : %lf \n", performance);



  time_gpu = time_end-time_start;


  if(matrix_compare(C,D,WC*HC)){
    printf("Multiplication correct\n");
  }
  else 
    printf("Multiplication incorrect\n");

  speed_up = (int)time_cpu/time_gpu;
  printf("Speed UP x%d\n",speed_up);

  memset(C,0,WC*HC*sizeof(float));

  printf("\n\nMultiplication with second optimisation\n");
  printf("-----------------------------------------\n");

  printf("\nSize Matrix %dx%d\n",WA,WB);

  // Start Multiplication GPU

  gettimeofday(&tv,NULL);
  time_start = tv.tv_sec+(tv.tv_usec/1000000.0);
  printf("Starting multiplication GPU ...");
  matrixMultiplication3(A,WA,HA,B,WB,HB,C,WC,HC);
  gettimeofday(&tv,NULL);
  time_end = tv.tv_sec+(tv.tv_usec/1000000.0);
  printf("[OK]\n");
  printf("Time : %f s\n",(float)(time_end-time_start));

  time_gpu = time_end-time_start;


  if(matrix_compare(C,D,WC*HC)){
    printf("Multiplication correct\n");
  }
  else 
    printf("Multiplication incorrect\n");

  speed_up = (int)time_cpu/time_gpu;
  printf("Speed UP x%d\n",speed_up);



  //  print_matrix(C,HC,WC);
  //  print_matrix(D,HC,WC);

  free(A);
  free(B);
  free(C);
  free(D);
}


void init_matrix(float * M,int col,int lin){
  int i = 0;
  for(i = 0; i < col * lin; ++i)
    M[i] = rand() / (float)RAND_MAX;
}



void matrixMultiplication(float* A,int wa,int ha,float* B,int wb,int hb,float* C,int wc ,int hc){
  
  float* gpu_A;
  float* gpu_B;
  float* gpu_C;
  unsigned int sizeA = sizeof(float) * wa * ha;
  unsigned int sizeB = sizeof(float) * wb * hb;
  unsigned int sizeC = sizeof(float) * wc * hc;
  hipMalloc((void**) &gpu_A, sizeA);
  hipMalloc((void**) &gpu_B, sizeB);
  hipMemcpy(gpu_A,A,sizeA,hipMemcpyHostToDevice);
  hipMemcpy(gpu_B,B,sizeB,hipMemcpyHostToDevice);
  hipMalloc((void**) &gpu_C, sizeC);

  dim3 threads(32,32);
  dim3 grid( wc / threads.x, hc / threads.y);
  hipDeviceSynchronize();
  matrixMul<<< grid, threads >>>(gpu_C,gpu_A,gpu_B,wa,wb);
  hipDeviceSynchronize();
  hipMemcpy(C,gpu_C,sizeC,hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  hipFree(gpu_A);
  hipFree(gpu_B);
  hipFree(gpu_C);
}


void matrixMultiplication2(float* A,int wa,int ha,float* B,int wb,int hb,float* C,int wc ,int hc){
  
  float* gpu_A;
  float* gpu_B;
  float* gpu_C;
  unsigned int sizeA = sizeof(float) * wa * ha;
  unsigned int sizeB = sizeof(float) * wb * hb;
  unsigned int sizeC = sizeof(float) * wc * hc;
  hipMalloc((void**) &gpu_A, sizeA);
  hipMalloc((void**) &gpu_B, sizeB);
  hipMemcpy(gpu_A,A,sizeA,hipMemcpyHostToDevice);
  hipMemcpy(gpu_B,B,sizeB,hipMemcpyHostToDevice);
  hipMalloc((void**) &gpu_C, sizeC);

  dim3 threads(BLOCK_SIZE,BLOCK_SIZE);
  dim3 grid( wc / threads.x, hc / threads.y);
  hipDeviceSynchronize();
  matrixMul2<<< grid, threads >>>(gpu_C,gpu_A,gpu_B,wa,wb);
  hipDeviceSynchronize();
  hipMemcpy(C,gpu_C,sizeC,hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  hipFree(gpu_A);
  hipFree(gpu_B);
  hipFree(gpu_C);
}



void matrixMultiplication3(float* A,int wa,int ha,float* B,int wb,int hb,float* C,int wc ,int hc){
  
  float* gpu_A;
  float* gpu_B;
  float* gpu_C;
  unsigned int sizeA = sizeof(float) * wa * ha;
  unsigned int sizeB = sizeof(float) * wb * hb;
  unsigned int sizeC = sizeof(float) * wc * hc;
  hipMalloc((void**) &gpu_A, sizeA);
  hipMalloc((void**) &gpu_B, sizeB);
  hipMemcpy(gpu_A,A,sizeA,hipMemcpyHostToDevice);
  hipMemcpy(gpu_B,B,sizeB,hipMemcpyHostToDevice);
  hipMalloc((void**) &gpu_C, sizeC);

  dim3 threads(16,16);
  dim3 grid( wc / threads.x, hc / threads.y);
  hipDeviceSynchronize();
  matrixMul3<<< grid, threads >>>(gpu_C,gpu_A,gpu_B,wa,wb);
  hipDeviceSynchronize();
  hipMemcpy(C,gpu_C,sizeC,hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  hipFree(gpu_A);
  hipFree(gpu_B);
  hipFree(gpu_C);
}

bool matrix_compare(float *a, float *b,int wc){
  int i = 0;
  for(i = 0;i < wc;++i){
    if( a[i] - b[i] > 0.1 || a[i] - b[i] < -0.1 ){
      return false;  
    }
  }
  return true;
}

void matrixMultiplicationOMP(float* A,int wa,int ha,float* B,int wb,int hb,float* C,int wc ,int hc){
  int i,j,k;
  int n = wa;
  omp_set_num_threads(8);

#pragma omp parallel for default(none) shared(A,B,C,wa,n) private(k,j)
  for (i=0; i<wa; ++i)
    {
       for(j=0; j<wa; ++j)       
	 {
            for (k=0; k<wa; ++k)
	      {
                 C[i*wa+j] += A[i*n+k] * B[k*n+j];
              }
         }
    }
}





void matrixMultiplicationSlow(float* A,int wa,int ha,float* B,int wb,int hb,float* C,int wc ,int hc){
  int n = wa;
  int i,j,k;
  for(i = 0 ; i < n ; ++i){
    for(j = 0 ; j < n ; ++j){
      for(k=0 ; k < n  ; ++k){
	C[i*n+j] = C[i*n+j] +  A[i*n+k]*B[k*n+j];
      }
    }
  }


}


void print_matrix(float* A,int lin,int col){
 printf("\n\nMatrix\n");
 for(int i = 0; i < lin*col; i++)
    {
      printf("%f ", A[i]);
      if(((i + 1) % col) == 0)
	printf("\n");
    }
 printf("\n");
 
}


void print_information(){



  struct hipDeviceProp_t * cudaInfos = (struct hipDeviceProp_t *)  malloc(sizeof(struct hipDeviceProp_t));
  int number_of_device;
  hipGetDeviceCount(&number_of_device);
  printf("System Informations\n");
  printf("-------------------\n\n");
  printf("Number of devices : %d\n",number_of_device);
  int device;
  hipGetDevice(&device);
  hipGetDeviceProperties(cudaInfos,0);
  
  
  printf("Device Informations\n");
  printf("  Name : %s\n",cudaInfos->name);
  printf("  Memory : %li\n",cudaInfos->totalGlobalMem);
  printf("  WarpSize : %d\n",cudaInfos->warpSize);
  printf("  Max Threads Per Block : %d\n",cudaInfos->maxThreadsPerBlock);
  printf("  Multi processor count : %d\n",cudaInfos->multiProcessorCount);


  hipDeviceSynchronize();
  //  free(cudaInfos);
}
void 
perf(perf_t * p) {
  gettimeofday(p, NULL);  
}

void 
perf_diff(const perf_t * begin, perf_t * end) {
  end->tv_sec = end->tv_sec - begin->tv_sec;
  end->tv_usec = end->tv_usec - begin->tv_usec;
  if (end->tv_usec < 0) {
    (end->tv_sec)--;
    end->tv_usec += 1000000;
  }
}

void
perf_printh(const perf_t * p) {
  long m = p->tv_sec / 60;
  long s = p->tv_sec - m*60;
  long ms = p->tv_usec / 1000;
  long micros = p->tv_usec - ms*1000;

  //  printf("%ld sec %ld usec\n", p->tv_sec, p->tv_usec);
  printf("%ld:%ld:%ld:%ld\n",m,s,ms,micros);
}

void
perf_printmicro(const perf_t * p) {
  printf("%ld\n",p->tv_usec + ( p->tv_sec * 1000000) );
}


double
perf_mflops(const perf_t * p, const long nb_op) {
  return (double)nb_op / (p->tv_sec * 1000000 + p->tv_usec);
}
